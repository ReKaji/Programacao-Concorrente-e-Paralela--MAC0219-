/* 
Este arquivo implementa a Tarefa 3 do EP2 da disciplina MAC0219
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define WALL_TEMP 20.0
#define FIREPLACE_TEMP 100.0
#define BODY_TEMPERATURE 37.0
#define FIREPLACE_START 3
#define FIREPLACE_END 7
#define ROOM_SIZE 10

void initialize(double *h,double *seqh,  int n)
{
    int fireplace_start = (FIREPLACE_START * n) / ROOM_SIZE;
    int fireplace_end = (FIREPLACE_END * n) / ROOM_SIZE;

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (i == 0 || i == n - 1 || j == 0 || j == n - 1)
            {
                h[i * n + j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
                seqh[i * n + j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
            }
            else if((i>=n/2-n/10 && i<=n/2+n/10) && 
                    (j>=n/2-n/10 && j<=n/2+n/10)){
                h[i*n+j]=BODY_TEMPERATURE;
                seqh[i*n+j]=BODY_TEMPERATURE;

            }
            else
            {
                h[i * n +j] = 0.0;
                seqh[i * n +j] = 0.0;

            }
        }
    }
}


bool compara_cpu_gpu(double *h, double *seqh, int n){
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (fabs(h[i*n+j]-seqh[i*n+j])>0.5) 
                return false;
              
        }
        
    }
    return true;
}

__global__ void jacobi_iteration(double *h, double *g, int n) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int stride_x = gridDim.x * blockDim.x;
    int stride_y = gridDim.y * blockDim.y; 

    for (int row = i; row < n - 1; row += stride_x) {
        for (int col = j; col < n - 1; col += stride_y) {
 
            if (row > 0 && row < n -1 && col > 0 && col < n-1 && (!(row >= n / 2 - n / 10 && row <= n / 2 + n / 10 && 
                      col >= n / 2 - n / 10 && col <= n / 2 + n / 10))) {
                g[row * n + col] = 0.25 * (h[(row - 1) * n + col] +
                                           h[(row + 1) * n + col] +
                                           h[row * n + col - 1] +
                                           h[row * n + col + 1]);
            }
        }
    }
    __syncthreads();
    for (int row = i; row < n -1; row += stride_x) {
        for (int col = j; col < n -1; col += stride_y) {
            if (row > 0 && row < n -1 && col > 0 && col < n-1 && (!(row >= n / 2 - n / 10 && row <= n / 2 + n / 10 && 
                      col >= n / 2 - n / 10 && col <= n / 2 + n / 10)))
                h[row*n+col]=g[row*n+col];
       }}
    __syncthreads();
}

void seq_jacobi_iteration(double *h, double *g, int n, int iter_limit)
{
    for (int iter = 0; iter < iter_limit; iter++)
    {
        for (int i = 1; i < n - 1; i++)
        {
            for (int j = 1; j < n - 1; j++)
            {
                if (i > 0 && i < n -1 && j > 0 && j < n-1 && (!(i >= n / 2 - n / 10 && i <= n / 2 + n / 10 && 
                      j >= n / 2 - n / 10 && j <= n / 2 + n / 10)))
                    g[i * n +j] = 0.25 * (h[(i-1)*n+ j] + h[(i + 1)*n+j] + h[i * n + j - 1] + h[i * n + j + 1]);
            }
        }
        for (int i = 1; i < n - 1; i++)
        {
            for (int j = 1; j < n - 1; j++)
            {
                if (i > 0 && i < n -1 && j > 0 && j < n-1 && (!(i >= n / 2 - n / 10 && i <= n / 2 + n / 10 && 
                      j >= n / 2 - n / 10 && j <= n / 2 + n / 10)))
                    h[i * n + j] = g[i*n+j];
            }
        }
    }
}
double calculate_elapsed_time(struct timespec start, struct timespec end)
{
    double start_sec = (double)start.tv_sec * 1e9 + (double)start.tv_nsec;
    double end_sec = (double)end.tv_sec * 1e9 + (double)end.tv_nsec;
    return (end_sec - start_sec) / 1e9;
}

void save_to_file(double *h, int n)
{
    FILE *file = fopen("room.txt", "w");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            fprintf(file, "%lf ", h[i*n+j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

int main(int argc, char *argv[])
{
    if (argc < 5)
    {
        fprintf(stderr, "Uso: %s <número de pontos> <limite de iterações> <t> <b>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);
    int t= atoi(argv[3]);
    int b=atoi (argv[4]);

    double *h = (double *)malloc(n * n * sizeof(double));
    double *g = (double *)malloc(n * n * sizeof(double));

    double *seqh = (double *)malloc(n * n * sizeof(double));
    double *seqg = (double *)malloc(n * n * sizeof(double));
    

    double *dh, *dg;

    
    hipMalloc((void**)&dh, sizeof(double) * n * n);
    hipMalloc((void**)&dg, sizeof(double) * n * n);
    
    



    if (h == NULL || g == NULL)
    {
        fprintf(stderr, "Erro ao alocar memória para h ou g\n");
        exit(EXIT_FAILURE);
    }


   
    initialize(h, seqh,n);

    float tempo_host_device=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpy (dg, g,n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy (dh, h, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tempo_host_device, start, stop);
    tempo_host_device/=1000;
    printf("Tempo de hipMemcpyHostToDevice de g e h : %.9f segundos\n", tempo_host_device);

    int raiz_t= (int)sqrt(t);
    int raiz_b=(int)sqrt(b);
    int tam_bloco1;
    int tam_bloco2;
    int tam_thread1;
    int tam_thread2;

    if (raiz_t * raiz_t == t){
        tam_thread1=  raiz_t;
        tam_thread2= raiz_t;
    }
    else
        {
            t=t/2;
            int raiz= (int)sqrt(t);
            tam_thread1= raiz;
            tam_thread2=2* raiz;
        }

    if (raiz_b * raiz_b == b){
        tam_bloco1=  raiz_b;
        tam_bloco2= raiz_b;
    }
    else
        {
            b=b/2;
            int raiz_b= (int)sqrt(b);
            tam_bloco1= raiz_b;
            tam_bloco2=2*raiz_b;
        }




    dim3 threads (tam_thread1,tam_thread2);
    dim3 blocks(tam_bloco1,tam_bloco2);

    float tempo_cuda = 0;
    float tempo_tarefa=0;
    for (int i=0; i< iter_limit;i++){
        
        hipEventRecord(start);
        jacobi_iteration<<<blocks,threads>>>(dh, dg, n);
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&tempo_tarefa, start, stop);
        tempo_cuda+=tempo_tarefa;
    }
    tempo_cuda/=1000;
    printf("Tempo de execução versão CUDA: %.9f segundos\n", tempo_cuda);
    
    float tempo_device_host=0;

    hipEventRecord(start);
    hipMemcpy (h, dh, n*n*sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tempo_device_host, start, stop);
    tempo_device_host/=1000;
    printf("Tempo de hipMemcpyDeviceToHost de h: %.9f segundos\n", tempo_device_host);
   
    struct timespec start1, end1;

    clock_gettime(CLOCK_MONOTONIC, &start1);
    seq_jacobi_iteration(seqh, seqg, n, iter_limit);
    clock_gettime(CLOCK_MONOTONIC, &end1);

    save_to_file(h, n);

    double elapsed_time = calculate_elapsed_time(start1, end1);
    printf("Tempo de execução versão sequncial: %.9f segundos\n", elapsed_time);
    if (compara_cpu_gpu(h, seqh, n)) 
        printf("As versões da GPU e CPU produzem o mesmo resultado\n");
    else 
        printf("As versões da GPU e CPU NÃO produzem o mesmo resultado\n");


    free(h);   
    free(g);
    free(seqg);
    free(seqh);
    hipFree(dg);
    hipFree(dh);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}